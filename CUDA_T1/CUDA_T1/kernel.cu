#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <math.h>
#include <stdlib.h>
#include <windows.h>
#include <time.h>
#include <process.h>

int threadsPerBlock = 1024;

double total_Time_CPU = 0;
double total_Time_GPU = 0;
LARGE_INTEGER beginClock, endClock, clockFreq;
LARGE_INTEGER tot_beginClock, tot_endClock, tot_clockFreq;

typedef struct RGB {
	unsigned char red;
	unsigned char green;
	unsigned char blue;
}RGB;
RGB * b_pix;	// 5 X 5개 복붙한 이미지 픽셀
RGB * pix;
BITMAPFILEHEADER bfh;
BITMAPINFOHEADER bih;

// 이미지 정보를 다루기 위해 사용하는 변수
int bpl, b_bpl;
int width, height, b_width, b_height;
int pix_size;
int b_pix_size;	// 5 X 5개 만큼 복붙한 이미지 사이즈
//unsigned char* pix; // 원본 이미지
//unsigned char* pix_out; // GPU 연산결과 이미지

void GraphicInfo();				// 현재 장착된 그래픽카드의 정보를 불러온다
char str[100];
void Fwrite(char * fn);		// 연산된 픽셀값을 bmp파일로 저장한다

hipError_t addWithCuda(RGB* a, int size);

__global__ void addKernel(RGB* a, RGB* d_pix, const int width, const int b_width, const int height)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
	int px = (i % b_width) % width;
	int py = (i % (b_width * height)) / b_width;
	int i2 = px + width * py;
	a[i].red = d_pix[i2].red;
	a[i].green = d_pix[i2].green;
	a[i].blue = d_pix[i2].blue;
}

int main()
{
	GraphicInfo();
	FILE * fp;
	fp = fopen("323test1.bmp", "rb");

	if (fp == NULL)
	{
		printf("File Not Found!!\n");
		return 0;
	}
	// 파일헤더, 정보헤더 읽어들인다
	fread(&bfh, sizeof(bfh), 1, fp);
	fread(&bih, sizeof(bih), 1, fp);

	width = bih.biWidth;
	height = bih.biHeight;
	b_width = width * 5;
	b_height = height * 5;

	// BPL을 맞춰주기 위해서 픽셀데이터의 사이즈를 4의 배수로 조정
	bpl = (width + 3) / 4 * 4;
	b_bpl = (b_width + 3) / 4 * 4;

	// 이미지 사이즈 정보
	pix_size = bih.biSizeImage / 3;
	b_pix_size = bih.biSizeImage * 25 / 3;
	printf("Image size : %d X %d\n", width, height);
	printf("Memory size : %d byte\n", pix_size * 3);
	printf("5 X 5 Image size : %d X %d\n", b_width, b_height);
	printf("Memory size : %d byte\n", b_pix_size * 3);

	// 원본 이미지 데이터 읽어 들인다
	pix = (RGB *)calloc(bpl * height, sizeof(RGB));
	fread(pix, sizeof(RGB), pix_size, fp);

	b_pix = (RGB *)calloc(b_pix_size, sizeof(RGB));

	QueryPerformanceFrequency(&tot_clockFreq);	// 시간을 측정하기위한 준비

	/*
	QueryPerformanceCounter(&tot_beginClock); // CPU 시간측정 시작
	for (int i = 0; i < pix_size; i++)
	{
		pix[i] += 5;
		//printf("%d\n", pix[i]);
	}
	QueryPerformanceCounter(&tot_endClock); // CPU 시간측정 종료
	total_Time_CPU = (double)(tot_endClock.QuadPart - tot_beginClock.QuadPart) / tot_clockFreq.QuadPart;

	sprintf(str, "323test1.bmp_CPU.bmp");
	Fwrite(str);
	*/
	
	QueryPerformanceCounter(&tot_beginClock); // GPU 시간측정 시작
	// Add vectors in parallel.
	hipError_t cudaStatus = addWithCuda(b_pix, threadsPerBlock);
	QueryPerformanceCounter(&tot_endClock); // GPU 시간측정 종료
	total_Time_GPU = (double)(tot_endClock.QuadPart - tot_beginClock.QuadPart) / tot_clockFreq.QuadPart;

	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addWithCuda failed!");
		return 1;
	}
	
	// hipDeviceReset must be called before exiting in order for profiling and
	// tracing tools such as Nsight and Visual Profiler to show complete traces.
	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceReset failed!");
		return 1;
	}

	printf("CPU 실행시간 : %f\nGPU 실행시간 : %f\n",
		total_Time_CPU * 1000, total_Time_GPU * 1000);

	sprintf(str, "323test1_GPU.bmp");
	Fwrite(str);
	free(pix);
	free(b_pix);
	fclose(fp);

	return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(RGB* a, int thread)
{
    RGB * dev_a = 0;
	RGB * d_pix = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (one input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_a, b_pix_size * sizeof(RGB));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

	cudaStatus = hipMalloc((void**)&d_pix, pix_size * sizeof(RGB));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}


    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, b_pix_size * sizeof(RGB), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

	cudaStatus = hipMemcpy(d_pix, pix, pix_size * sizeof(RGB), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

    // Launch a kernel on the GPU with one thread for each element.
	// 함수명<<<블록 수, 스레드 수>>>(매개변수);
    addKernel<<< b_pix_size/thread + 1, thread >>>(dev_a, d_pix, width, b_width, height);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(a, dev_a, b_pix_size * sizeof(RGB), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }



Error:
    hipFree(dev_a);
	hipFree(d_pix);

    return cudaStatus;
}
void GraphicInfo()
{
	hipDeviceProp_t  prop;

	int count;
	hipGetDeviceCount(&count);

	for (int i = 0; i < count; i++) {
		hipGetDeviceProperties(&prop, i);
		printf("   --- General Information for device %d ---\n", i);
		printf("Name:  %s\n", prop.name);
		printf("Compute capability:  %d.%d\n", prop.major, prop.minor);
		printf("Clock rate:  %d\n", prop.clockRate);
		printf("Device copy overlap:  ");
		if (prop.deviceOverlap)
			printf("Enabled\n");
		else
			printf("Disabled\n");
		printf("Kernel execution timeout :  ");
		if (prop.kernelExecTimeoutEnabled)
			printf("Enabled\n");
		else
			printf("Disabled\n");
		printf("\n");

		printf("   --- Memory Information for device %d ---\n", i);
		printf("Total global mem:  %ld\n", prop.totalGlobalMem);
		printf("Total constant Mem:  %ld\n", prop.totalConstMem);
		printf("Max mem pitch:  %ld\n", prop.memPitch);
		printf("Texture Alignment:  %ld\n", prop.textureAlignment);
		printf("\n");

		printf("   --- MP Information for device %d ---\n", i);
		printf("Multiprocessor count:  %d\n", prop.multiProcessorCount);
		printf("Shared mem per mp:  %ld\n", prop.sharedMemPerBlock);
		printf("Registers per mp:  %d\n", prop.regsPerBlock);
		printf("Threads in warp:  %d\n", prop.warpSize);
		printf("Max threads per block:  %d\n", prop.maxThreadsPerBlock);
		printf("Max thread dimensions:  (%d, %d, %d)\n", prop.maxThreadsDim[0], prop.maxThreadsDim[1], prop.maxThreadsDim[2]);
		printf("Max grid dimensions:  (%d, %d, %d)\n", prop.maxGridSize[0], prop.maxGridSize[1], prop.maxGridSize[2]);
		printf("--------------------------------------\n");
		printf("\n");
	}

}

// 데이터 픽셀값을 bmp파일로 쓴다.
void Fwrite(char * fn)
{
	FILE * fp2 = fopen(fn, "wb");
	bih.biWidth *= 5;
	bih.biHeight *= 5;
	bih.biSizeImage *= 25;

	fwrite(&bfh, sizeof(bfh), 1, fp2);
	fwrite(&bih, sizeof(bih), 1, fp2);
	fwrite(b_pix, sizeof(RGB), b_pix_size, fp2);
	fclose(fp2);
}