#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <math.h>
#include <stdlib.h>
#include <windows.h>
#include <time.h>
#include <process.h>

// 마스크값 배열
RGBTRIPLE Mask[25];

int threadsPerBlock = 1024;

double total_Time_CPU = 0;
double total_Time_GPU = 0;
LARGE_INTEGER beginClock, endClock, clockFreq;
LARGE_INTEGER tot_beginClock, tot_endClock, tot_clockFreq;

//BYTE * pix;
//BYTE * b_pix;
RGBTRIPLE * pix;
RGBTRIPLE * b_pix;
BITMAPFILEHEADER bfh;
BITMAPINFOHEADER bih;
BITMAPFILEHEADER b_bfh;
BITMAPINFOHEADER b_bih;

// 이미지 정보를 다루기 위해 사용하는 변수
int channel = 5;
int bpl, b_bpl;
int bpl_size, b_bpl_size;
int width, height, b_width, b_height;
int pix_size;
int b_pix_size;	// 5 X 5개 만큼 복붙한 이미지 사이즈
int pad, b_pad;		// 패딩 메모리
//unsigned char* pix; // 원본 이미지
//unsigned char* pix_out; // GPU 연산결과 이미지
BYTE * trash;

void GraphicInfo();				// 현재 장착된 그래픽카드의 정보를 불러온다
char str[100];
char str_Extend[100];
void Fwrite_Extend(char * fn);		// 연산된 픽셀값을 bmp파일로 저장한다
void Fwrite(char * fn);
void Draw();					// pix 데이터를 화면으로 출력
void b_Draw();					// b_pix 데이터를 화면으로 출력
hipError_t extendWithCuda(RGBTRIPLE* b_pix, int size);

__global__ void extendKernel(RGBTRIPLE* d_b_pix, RGBTRIPLE* d_pix, RGBTRIPLE* mask, const int width, const int b_width, const int height)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
	int px = i % width;
	int py = (i % (b_width * height)) / b_width;
	int ip = px + py * width;
	int mx = (i % b_width) / width;
	int my = i / (b_width * height);
	int im = mx + my * 5;

	d_b_pix[i].rgbtBlue = d_pix[ip].rgbtBlue;
	d_b_pix[i].rgbtGreen = d_pix[ip].rgbtGreen;
	d_b_pix[i].rgbtRed = d_pix[ip].rgbtRed;

	d_b_pix[i].rgbtBlue += mask[im].rgbtBlue;
	d_b_pix[i].rgbtGreen += mask[im].rgbtGreen;
	d_b_pix[i].rgbtRed += mask[im].rgbtRed;

	if (d_b_pix[i].rgbtBlue > 255)
		d_b_pix[i].rgbtBlue = 255;
	if (d_b_pix[i].rgbtBlue < 0)
		d_b_pix[i].rgbtBlue = 0;
	if (d_b_pix[i].rgbtGreen > 255)
		d_b_pix[i].rgbtGreen = 255;
	if (d_b_pix[i].rgbtGreen < 0)
		d_b_pix[i].rgbtGreen = 0;
	if (d_b_pix[i].rgbtRed > 255)
		d_b_pix[i].rgbtRed = 255;
	if (d_b_pix[i].rgbtRed < 0)
		d_b_pix[i].rgbtRed = 0;

	//d_b_pix[i] = d_pix[i2];
}

int main()
{
	for (int i = 0; i < 25; i++)
	{
		Mask[i].rgbtRed = i * 37;
		Mask[i].rgbtGreen = 50;
		Mask[i].rgbtBlue = i * 23;
	}

	GraphicInfo();
	FILE * fp;
	
	//fp = fopen("test3.bmp", "rb");
	//fp = fopen("lenna_406.bmp", "rb");
	//fp = fopen("323test5.bmp", "rb");
	fp = fopen("input.bmp", "rb");

	if (fp == NULL)
	{
		printf("File Not Found!!\n");
		system("pause");
		return 0;
	}
	// 파일헤더, 정보헤더 읽어들인다
	fread(&bfh, sizeof(bfh), 1, fp);
	fread(&bih, sizeof(bih), 1, fp);

	width = bih.biWidth;
	height = bih.biHeight;
	b_width = width * channel;
	b_height = height * channel;

	
	// BPL을 맞춰주기 위해서 픽셀데이터의 메모리를 4의 배수로 조정
	bpl = (width * 3 + 3) / 4 * 4;
	b_bpl = (b_width * 3 + 3) / 4 * 4;

	// 패딩 값 계산
	pad = bpl - width * 3;
	b_pad = b_bpl - b_width * 3;

	// BPL을 맞춘 메모리 사이즈
	bpl_size = bpl * height;
	b_bpl_size = b_bpl * b_height;

	// 순수 이미지 사이즈
	pix_size = width * height;
	b_pix_size = b_width * b_height;

	printf("Image size : %d X %d\n", width, height);
	printf("Memory size : %d byte\n", bpl_size);
	printf("%d X %d Image size : %d X %d\n", channel, channel, b_width, b_height);
	printf("%d X %d Memory size : %d byte\n", channel, channel, b_bpl_size);

	// 쓰레기 값
	trash = (BYTE *)calloc(b_pad, sizeof(BYTE));
	// 원본 이미지 데이터
	pix = (RGBTRIPLE *)calloc(pix_size, sizeof(RGBTRIPLE));
	for (int i = 0; i < height; i++)
	{
		fread(pix + (i * width), sizeof(RGBTRIPLE), width, fp);
		fread(&trash, sizeof(BYTE), pad, fp);
	}

	// 5 X 5 이미지 데이터
	b_pix = (RGBTRIPLE *)calloc(b_pix_size, sizeof(RGBTRIPLE));

	/*
	for(int i = 0; i < 1000; i++)
	{
		Draw();
	}
	*/

	QueryPerformanceFrequency(&tot_clockFreq);	// 시간을 측정하기위한 준비

	QueryPerformanceCounter(&tot_beginClock); // CPU 시간측정 시작
	for (int i = 0; i < b_pix_size; i++)
	{
		int px = i % width;
		int py = (i % (b_width * height)) / b_width;
		int ip = px + py * width;
		int mx = (i % b_width) / width;
		int my = i / (b_width * height);
		int im = mx + my * 5;

		b_pix[i].rgbtBlue = pix[ip].rgbtBlue;
		b_pix[i].rgbtGreen = pix[ip].rgbtGreen;
		b_pix[i].rgbtRed = pix[ip].rgbtRed;

		b_pix[i].rgbtBlue += Mask[im].rgbtBlue;
		b_pix[i].rgbtGreen += Mask[im].rgbtGreen;
		b_pix[i].rgbtRed += Mask[im].rgbtRed;

		if (b_pix[i].rgbtBlue > 255)
			b_pix[i].rgbtBlue = 255;
		if (b_pix[i].rgbtBlue < 0)
			b_pix[i].rgbtBlue = 0;
		if (b_pix[i].rgbtGreen > 255)
			b_pix[i].rgbtGreen = 255;
		if (b_pix[i].rgbtGreen < 0)
			b_pix[i].rgbtGreen = 0;
		if (b_pix[i].rgbtRed > 255)
			b_pix[i].rgbtRed = 255;
		if (b_pix[i].rgbtRed < 0)
			b_pix[i].rgbtRed = 0;
	}
	QueryPerformanceCounter(&tot_endClock); // CPU 시간측정 종료
	total_Time_CPU = (double)(tot_endClock.QuadPart - tot_beginClock.QuadPart) / tot_clockFreq.QuadPart;

	//sprintf(str_Extend, "323test5_Extend_CPU.bmp");
	sprintf(str_Extend, "output_CPU.bmp");

	//Fwrite(str);
	Fwrite_Extend(str_Extend);

	memset(b_pix, 0, sizeof(unsigned char) * b_pix_size);


	QueryPerformanceCounter(&tot_beginClock); // GPU 시간측정 시작
	// Add vectors in parallel.
	hipError_t cudaStatus = extendWithCuda(b_pix, threadsPerBlock);
	QueryPerformanceCounter(&tot_endClock); // GPU 시간측정 종료
	total_Time_GPU = (double)(tot_endClock.QuadPart - tot_beginClock.QuadPart) / tot_clockFreq.QuadPart;

	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "extendWithCuda failed!");
		system("pause");
		return 1;
	}
	
	// hipDeviceReset must be called before exiting in order for profiling and
	// tracing tools such as Nsight and Visual Profiler to show complete traces.
	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceReset failed!");
		system("pause");
		return 1;
	}
	printf("CPU 실행시간 : %f\nGPU 실행시간 : %f\n",
		total_Time_CPU * 1000, total_Time_GPU * 1000);

	printf("CPU / GPU = %lf\n",
		total_Time_CPU / total_Time_GPU);

	//sprintf(str, "test_GPU.bmp");
	//sprintf(str_Extend, "test3_Extend.bmp");
	//sprintf(str_Extend, "lenna_406_Extend.bmp");
	//sprintf(str_Extend, "323test5_Extend_GPU.bmp");
	sprintf(str_Extend, "output_GPU.bmp");

	//Fwrite(str);
	Fwrite_Extend(str_Extend);

	/*
	for (int i = 0; i < 1000; i++)
	{
		b_Draw();
	}
	*/


	free(pix);
	free(b_pix);
	free(trash);
	fclose(fp);

	system("pause");
	return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t extendWithCuda(RGBTRIPLE* b_pix, int thread)
{
	RGBTRIPLE * d_b_pix = 0;
	RGBTRIPLE * d_pix = 0;
	RGBTRIPLE * mask = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&d_b_pix, b_pix_size * sizeof(RGBTRIPLE));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

	cudaStatus = hipMalloc((void**)&d_pix, pix_size * sizeof(RGBTRIPLE));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&mask, 25 * sizeof(RGBTRIPLE));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

    // Copy input vectors from host memory to GPU buffers.
	cudaStatus = hipMemcpy(d_pix, pix, pix_size * sizeof(RGBTRIPLE), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	cudaStatus = hipMemcpy(mask, Mask, 25 * sizeof(RGBTRIPLE), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

    // Launch a kernel on the GPU with one thread for each element.
	// 함수명<<<블록 수, 스레드 수>>>(매개변수);
    extendKernel<<< (b_pix_size + thread - 1) / thread, thread >>>(d_b_pix, d_pix, mask, width, b_width, height);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(b_pix, d_b_pix, b_pix_size * sizeof(RGBTRIPLE), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }



Error:
    hipFree(d_b_pix);
	hipFree(d_pix);
	hipFree(mask);

    return cudaStatus;
}
void GraphicInfo()
{
	hipDeviceProp_t  prop;

	int count;
	hipGetDeviceCount(&count);

	for (int i = 0; i < count; i++) {
		hipGetDeviceProperties(&prop, i);
		printf("   --- General Information for device %d ---\n", i);
		printf("Name:  %s\n", prop.name);
		printf("Compute capability:  %d.%d\n", prop.major, prop.minor);
		printf("Clock rate:  %d\n", prop.clockRate);
		printf("Device copy overlap:  ");
		if (prop.deviceOverlap)
			printf("Enabled\n");
		else
			printf("Disabled\n");
		printf("Kernel execution timeout :  ");
		if (prop.kernelExecTimeoutEnabled)
			printf("Enabled\n");
		else
			printf("Disabled\n");
		printf("\n");

		printf("   --- Memory Information for device %d ---\n", i);
		printf("Total global mem:  %ld\n", prop.totalGlobalMem);
		printf("Total constant Mem:  %ld\n", prop.totalConstMem);
		printf("Max mem pitch:  %ld\n", prop.memPitch);
		printf("Texture Alignment:  %ld\n", prop.textureAlignment);
		printf("\n");

		printf("   --- MP Information for device %d ---\n", i);
		printf("Multiprocessor count:  %d\n", prop.multiProcessorCount);
		printf("Shared mem per mp:  %ld\n", prop.sharedMemPerBlock);
		printf("Registers per mp:  %d\n", prop.regsPerBlock);
		printf("Threads in warp:  %d\n", prop.warpSize);
		printf("Max threads per block:  %d\n", prop.maxThreadsPerBlock);
		printf("Max thread dimensions:  (%d, %d, %d)\n", prop.maxThreadsDim[0], prop.maxThreadsDim[1], prop.maxThreadsDim[2]);
		printf("Max grid dimensions:  (%d, %d, %d)\n", prop.maxGridSize[0], prop.maxGridSize[1], prop.maxGridSize[2]);
		printf("--------------------------------------\n");
		printf("\n");
	}

}

void Fwrite(char * fn)
{
	FILE * fp2 = fopen(fn, "wb");
	fwrite(&bfh, sizeof(bfh), 1, fp2);
	fwrite(&bih, sizeof(bih), 1, fp2);

	for (int i = 0; i < height; i++)
	{
		fwrite(pix + (i * width * 3), sizeof(BYTE), width * 3, fp2);
		fwrite(&trash, sizeof(BYTE), pad, fp2);
	}

	fclose(fp2);
}

// 데이터 픽셀값을 bmp파일로 쓴다.
void Fwrite_Extend(char * fn)
{
	FILE * fp2 = fopen(fn, "wb");
	b_bfh = bfh;
	b_bih = bih;
	b_bih.biWidth = b_width;
	b_bih.biHeight = b_height;
	b_bih.biSizeImage = b_bpl_size;
	b_bfh.bfSize = b_bih.biSizeImage + sizeof(BITMAPFILEHEADER) + sizeof(BITMAPINFOHEADER);

	fwrite(&b_bfh, sizeof(bfh), 1, fp2);
	fwrite(&b_bih, sizeof(bih), 1, fp2);
	
	for (int i = 0; i < b_height; i++)
	{
		fwrite(b_pix + (i * b_width), sizeof(RGBTRIPLE), b_width, fp2);
		fwrite(&trash, sizeof(BYTE), b_pad, fp2);
	}
	
	fclose(fp2);
}
// 연산된 RGB 값을 화면에 출력시킨다.
void Draw()
{
	HDC hdc;
	hdc = GetDC(NULL);

	SetDIBitsToDevice(hdc, 0, 0, bpl, height, 0, 0, 0, height,
		(BYTE *)pix, (const BITMAPINFO *)&bih, DIB_RGB_COLORS);

	ReleaseDC(NULL, hdc);
}
void b_Draw()
{
	HDC hdc;
	hdc = GetDC(NULL);

	SetDIBitsToDevice(hdc, 0, 0, b_width, height * 2, 0, 0, 0, height * 2,
		(BYTE *)b_pix, (const BITMAPINFO *)&b_bih, DIB_RGB_COLORS);

	ReleaseDC(NULL, hdc);
}

